#include "hip/hip_runtime.h"
/*
The following file is a CUDA kernel that we used to find mineshaft carver seeds that generate
a long corridor with a bunch of spider spawners. It exploits a weakness in Java Random that
allows iteration over states satisfying certain PRNG outpu conditions. In this case, the
condition is that a mineshaft generates, which is equivalent to
    rand.nextDouble() < 0.004
After some transformations, we obtain a direct way to iterate PRNG states such that the above
condition always evaluates to true.
With this optimization, the entire kernel takes around half a minute to run.
*/


#include "jrand/jrand.cuh"
#include "cudaSetup.cuh"
#include <cmath>

constexpr double MAX_VALUE = 0.004 * (double)(1ULL << 53u);
constexpr uint64_t MAX_UPPER_26 = 268435;
constexpr uint64_t MAX_TID = MAX_UPPER_26 * (1ULL << 22u);
constexpr uint64_t RUN_SIZE = 1ULL << 31;
constexpr int RUNS = (MAX_TID + RUN_SIZE - 1) / RUN_SIZE;


__global__ void getMineshaftCarvers(uint64_t offset) {
	const uint64_t tid = offset + (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= MAX_TID) return;

	const uint64_t lower22 = tid & 0x3FFFFFULL;
	const uint64_t upper26 = tid >> 22;
	const uint64_t initialSeed = (upper26 << 22) | lower22;

	uint64_t state = initialSeed;
	advance(&state); // skip call #2 in nextDouble

	advance2(&state);
	advance3(&state);

	int long_corrs = 0;
	for (int i = 0; i < 4; i++)
	{
		if (nextInt(&state, 100) >= 70) return; // only corridors
		int l = nextInt(&state, 3);
		if (l != 0) return; // short corridors only
		//if (l == 2) return; // dont want long ones
		//if (l == 1 && (++long_corrs >= 2)) return; // dont want long ones
		if (nextInt(&state, 3) == 0) return; // no rails
		if (nextInt(&state, 23) != 0) return; // cobwebs
		if (i != 3 && nextInt(&state, 4) > 1) return; // next piece same direction (north)
		advance(&state); // height diff unimportant
	}

	state = initialSeed;
	goBack(&state);
	state ^= JRAND_MULTIPLIER;
	const uint64_t carverSeed = (state & MASK48);

	// check trial chamber params (it will be in the same carver chunk!)
	state = carverSeed ^ JRAND_MULTIPLIER;
	int y = nextInt(&state, 21) - 41;
	//if (y < -37 || y > -35) return; // y coord bad
	int rot = nextInt(&state, 4);
	if (rot != 0) return; // rotation is not north

	printf("%llu  y = %d\n", carverSeed, y);
}

// ----------------------------------------------------------------

int findSpiderCorridors() {
	//printf("upper 26 bits at most: %llu\n", ((uint64_t)round(MAX_VALUE)) >> 27);
	//return 0;

	CHECK_ERR(hipSetDevice(0));

	for (int run = 0; run < RUNS; run++)
	{
		const int THREADS_PER_BLOCK = 512;
		const int N_BLOCKS = (int)((RUN_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
		getMineshaftCarvers <<< N_BLOCKS, THREADS_PER_BLOCK >>> (run * RUN_SIZE);
		CHECK_ERR(hipGetLastError());
		CHECK_ERR(hipDeviceSynchronize());

		//fprintf(stdout, "----- (stdout) Run %d/%d\n", run + 1, RUNS);
		fprintf(stderr, "----- (stderr) Run %d/%d\n", run + 1, RUNS);
	}

	return 0;
}

// ----------------------------------------------------------------

int main() {
	return findSpiderCorridors();
}